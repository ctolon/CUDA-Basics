#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void helloCuda(){

    printf("Hello World With CUDA!\n");
}

__global__ void helloCuda2() {

    printf("Hello World With CUDA 20 Times!\n");
}

__global__ void helloCuda3() {

    printf("Hello World With CUDA 16 Times!\n");
}

__global__ void helloCuda4() {

    printf("Hello World With CUDA Special\n");
}

int main(){

    int nx, ny;
    nx = 16; ny = 4;

    dim3 block(4);
    dim3 grid(4);

    dim3 block2(8,2);
    dim3 grid2(nx /block.x, ny/block.y);

    helloCuda << <1, 1 >> > ();
    helloCuda2 << <1, 20 >> > ();
    helloCuda3 << <grid,block >> > ();
    helloCuda4 << <grid2, block2 >> > ();
    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}

