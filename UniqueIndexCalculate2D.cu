#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


__global__ void unique_gid_calculation_2D(int* input) {
	int tid = threadIdx.x;
	int block_offset = blockIdx.x * blockDim.x;
	int row_offset = blockDim.x * gridDim.x * blockIdx.y;
	int gid = block_offset + row_offset + tid; // global index
	printf("blockIdx.x : %d, blockIdx.y : %d threadIdx : %d, gid : %d value : %d \n"
		,blockIdx.x, blockIdx.y, tid, gid, input[gid]);



}


int main() {

	int array_size = 16;
	int array_byte_size = sizeof(int) * array_size;
	int h_data[] = { 23,9,4,53,65,12,1,33,87,45,23,12,342,56,44,99 };
	printf("My Array List :");
	for (int i = 0; i < array_size; i++) {
		printf("%d ", h_data[i]);
	}
	printf("\n \n");

	int* d_data;
	hipMalloc((void**)&d_data, array_byte_size);
	hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

	dim3 block(4);
	dim3 grid(2,2);



	unique_gid_calculation_2D << < grid, block >> > (d_data);
	hipDeviceSynchronize();

	hipDeviceReset();
	return 0;
}

